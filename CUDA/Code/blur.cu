#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


static int const height = 521,
                 width  = 428,
                 maxLineLength = 200,
                 maxHeaderSize = 5,
                 maxX = width - 1,
                 maxY = height - 1,
                 arraySize = width * height * sizeof(int);


void readInputFile (int h_R[width][height], int h_G[width][height], int h_B[width][height], char header[maxHeaderSize][maxLineLength], int *headerSize)
{
    unsigned int h1, h2, h3;
    int x = 0, y = 0;
    char *sptr, line[maxLineLength];
    FILE *fp;
    fp = fopen("David.ps", "r");

    *headerSize = 0;
 
    while(! feof(fp))
    {
        fscanf(fp, "\n%[^\n]", line);
        if (*headerSize < 5) {
            strcpy((char *)header[(*headerSize)++], (char *)line);
        }
        else {
            for (sptr = &line[0]; *sptr != '\0'; sptr += 6) {
                sscanf(sptr,"%2x",&h1);
                sscanf(sptr+2,"%2x",&h2);
                sscanf(sptr+4,"%2x",&h3);
                
                if (x == width) {
                    x = 0;
                    y++;
                }
                if (y < height) {
                    h_R[x][y] = h1;
                    h_G[x][y] = h2;
                    h_B[x][y] = h3;
                }
                x++;
            }
        }
    }
    fclose(fp);
}


void writeOutputFile (int h_R[width][height], int h_G[width][height], int h_B[width][height], char header[maxHeaderSize][maxLineLength], int headerSize)
{
    int linelen = 12,
        charPos = 0;
    FILE *fout;

    fout= fopen("DavidBlur.ps", "w");
    for (int i = 0; i < headerSize; i++) fprintf(fout,"\n%s", header[i]);
    fprintf(fout,"\n");

    for(int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            fprintf(fout, "%02x%02x%02x", h_R[x][y], h_G[x][y], h_B[x][y]);
            if (++charPos == linelen) {
                fprintf(fout,"\n");
                charPos = 0;
            }
        }
    }
    fclose(fout);
}


void allocateDeviceMemory (int (**d_RIn)[width][height], int (**d_GIn)[width][height], int (**d_BIn)[width][height], int (**d_ROut)[width][height], int (**d_GOut)[width][height], int (**d_BOut)[width][height])
{
    hipMalloc(d_RIn, arraySize);
    hipMalloc(d_GIn, arraySize);
    hipMalloc(d_BIn, arraySize);
    hipMalloc(d_ROut, arraySize);
    hipMalloc(d_GOut, arraySize);
    hipMalloc(d_BOut, arraySize);
}


void freeDeviceMemory (int d_RIn[width][height], int d_GIn[width][height], int d_BIn[width][height], int d_ROut[width][height], int d_GOut[width][height], int d_BOut[width][height])
{
    hipFree(d_RIn);
    hipFree(d_GIn);
    hipFree(d_BIn);
    hipFree(d_ROut);
    hipFree(d_GOut);
    hipFree(d_BOut);
}


void copyMemoryToDevice (int h_R[width][height], int h_G[width][height], int h_B[width][height], int d_RIn[width][height], int d_GIn[width][height], int d_BIn[width][height])
{
    hipMemcpy(d_RIn, h_R, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_GIn, h_G, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_BIn, h_B, arraySize, hipMemcpyHostToDevice);
}


void copyMemoryFromDevice (int d_ROut[width][height], int d_GOut[width][height], int d_BOut[width][height], int h_R[width][height], int h_G[width][height], int h_B[width][height])
{
    hipMemcpy(h_R, d_ROut, arraySize, hipMemcpyDeviceToHost);
    hipMemcpy(h_G, d_GOut, arraySize, hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_BOut, arraySize, hipMemcpyDeviceToHost);
}


__global__
void blurKernel (int d_RIn[width][height], int d_GIn[width][height], int d_BIn[width][height], int d_ROut[width][height], int d_GOut[width][height], int d_BOut[width][height])
{
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x != 0 && x != maxX && y != 0 && y != maxY) {
        d_ROut[x][y] = (d_RIn[x+1][y] + d_RIn[x-1][y] + d_RIn[x][y+1] + d_RIn[x][y-1]) / 4;
        d_GOut[x][y] = (d_GIn[x+1][y] + d_GIn[x-1][y] + d_GIn[x][y+1] + d_GIn[x][y-1]) / 4;
        d_BOut[x][y] = (d_BIn[x+1][y] + d_BIn[x-1][y] + d_BIn[x][y+1] + d_BIn[x][y-1]) / 4;
    }
    else if (x == maxX && y != 0 && y != maxY) {
        d_ROut[x][y] = (                d_RIn[x-1][y] + d_RIn[x][y+1] + d_RIn[x][y-1]) / 3;
        d_GOut[x][y] = (                d_GIn[x-1][y] + d_GIn[x][y+1] + d_GIn[x][y-1]) / 3;
        d_BOut[x][y] = (                d_BIn[x-1][y] + d_BIn[x][y+1] + d_BIn[x][y-1]) / 3;
    }
    else if (x == 0 && y != 0 && y != maxY) {
        d_ROut[x][y] = (d_RIn[x+1][y]                 + d_RIn[x][y+1] + d_RIn[x][y-1]) / 3;
        d_GOut[x][y] = (d_GIn[x+1][y]                 + d_GIn[x][y+1] + d_GIn[x][y-1]) / 3;
        d_BOut[x][y] = (d_BIn[x+1][y]                 + d_BIn[x][y+1] + d_BIn[x][y-1]) / 3;
    }
    else if (y == maxY && x != 0 && x != maxX) {
        d_ROut[x][y] = (d_RIn[x+1][y] + d_RIn[x-1][y]                 + d_RIn[x][y-1]) / 3;
        d_GOut[x][y] = (d_GIn[x+1][y] + d_GIn[x-1][y]                 + d_GIn[x][y-1]) / 3;
        d_BOut[x][y] = (d_BIn[x+1][y] + d_BIn[x-1][y]                 + d_BIn[x][y-1]) / 3;
    }
    else if (y == 0 && x != 0 && x != maxX) {
        d_ROut[x][y] = (d_RIn[x+1][y] + d_RIn[x-1][y] + d_RIn[x][y+1]                ) / 3;
        d_GOut[x][y] = (d_GIn[x+1][y] + d_GIn[x-1][y] + d_GIn[x][y+1]                ) / 3;
        d_BOut[x][y] = (d_BIn[x+1][y] + d_BIn[x-1][y] + d_BIn[x][y+1]                ) / 3;
    }
    else if (x == maxX && y == 0) {
        d_ROut[x][y] = (                d_RIn[x-1][y] + d_RIn[x][y+1]                ) / 2;
        d_GOut[x][y] = (                d_GIn[x-1][y] + d_GIn[x][y+1]                ) / 2;
        d_BOut[x][y] = (                d_BIn[x-1][y] + d_BIn[x][y+1]                ) / 2;
    }
    else if (x == 0 && y == maxY) {
        d_ROut[x][y] = (d_RIn[x+1][y]                                 + d_RIn[x][y-1]) / 2;
        d_GOut[x][y] = (d_GIn[x+1][y]                                 + d_GIn[x][y-1]) / 2;
        d_BOut[x][y] = (d_BIn[x+1][y]                                 + d_BIn[x][y-1]) / 2;
    }
    else if (x == maxX && y == maxY) {
        d_ROut[x][y] = (                d_RIn[x-1][y]                 + d_RIn[x][y-1]) / 2;
        d_GOut[x][y] = (                d_GIn[x-1][y]                 + d_GIn[x][y-1]) / 2;
        d_BOut[x][y] = (                d_BIn[x-1][y]                 + d_BIn[x][y-1]) / 2;
    }
    else if (x == 0 && y == 0) {
        d_ROut[x][y] = (d_RIn[x+1][y]                 + d_RIn[x][y+1]                ) / 2;
        d_GOut[x][y] = (d_GIn[x+1][y]                 + d_GIn[x][y+1]                ) / 2;
        d_BOut[x][y] = (d_BIn[x+1][y]                 + d_BIn[x][y+1]                ) / 2;
    }
}


void outputTimingResults (struct timeval t1, struct timeval t2, struct timeval t3, struct timeval t4, struct timeval t5, struct timeval t6, struct timeval t7, struct timeval t8)
{
    // Convert times to seconds
    double t1_s = t1.tv_sec + t1.tv_usec / 1000000.0,
           t2_s = t2.tv_sec + t2.tv_usec / 1000000.0,
           t3_s = t3.tv_sec + t3.tv_usec / 1000000.0,
           t4_s = t4.tv_sec + t4.tv_usec / 1000000.0,
           t5_s = t5.tv_sec + t5.tv_usec / 1000000.0,
           t6_s = t6.tv_sec + t6.tv_usec / 1000000.0,
           t7_s = t7.tv_sec + t7.tv_usec / 1000000.0,
           t8_s = t8.tv_sec + t8.tv_usec / 1000000.0;

    // Calculate intervals between times
    double t1t2_s = t2_s - t1_s,
           t2t3_s = t3_s - t2_s,
           t3t4_s = t4_s - t3_s,
           t4t5_s = t5_s - t4_s,
           t5t6_s = t6_s - t5_s,
           t6t7_s = t7_s - t6_s,
           t7t8_s = t8_s - t7_s;

    // Print final timings
    printf("Read Input File: %f\n", t1t2_s);
    printf("Allocate Device Memory: %f\n", t2t3_s);
    printf("Copy Memory to Device: %f\n", t3t4_s);
    printf("Blur: %f\n", t4t5_s);
    printf("Copy Memory from Device: %f\n", t5t6_s);
    printf("Free Memory on Device: %f\n", t6t7_s);
    printf("Write Output File: %f\n", t7t8_s);
}


int main (int argc, const char * argv[])
{
    // Record the time at different points in execution
    struct timeval t1, t2, t3, t4, t5, t6, t7, t8;
    char header[maxHeaderSize][maxLineLength];
    int nblurs = atoi(argv[1]),
        gridWidth,
        gridHeight,
        headerSize,
        blockWidth,
        blockHeight,
        h_R[width][height],
        h_G[width][height],
        h_B[width][height],
        // Use pointers to allow swapping input and output arrays in-between blurs without moving memory around 
        (*swap)[width][height],
        (*d_RIn)[width][height],
        (*d_GIn)[width][height],
        (*d_BIn)[width][height],
        (*d_ROut)[width][height],
        (*d_GOut)[width][height],
        (*d_BOut)[width][height];
    
    gettimeofday(&t1, NULL);
    readInputFile(h_R, h_G, h_B, header, &headerSize);
    gettimeofday(&t2, NULL);
    allocateDeviceMemory(&d_RIn, &d_GIn, &d_BIn, &d_ROut, &d_GOut, &d_BOut);
    gettimeofday(&t3, NULL);
    copyMemoryToDevice(h_R, h_G, h_B, *d_RIn, *d_GIn, *d_BIn);
    gettimeofday(&t4, NULL);
    
    blockWidth  = 16;
    blockHeight = 16;
    gridWidth  = ceil((double)width  / blockWidth);
    gridHeight = ceil((double)height / blockHeight);
    dim3 dimGrid(gridWidth,   gridHeight,  1);
    dim3 dimBlock(blockWidth, blockHeight, 1);
    
    // nblurs passed as commandline argument to avoid interfering with timing
    // nblurs = 10;
    // printf("\nGive the number of times to blur the image\n");
    // int icheck = scanf("%d", &nblurs);

    // Do first blur without swapping output and input pointers
    blurKernel<<<dimGrid, dimBlock>>>(*d_RIn, *d_GIn, *d_BIn, *d_ROut, *d_GOut, *d_BOut);
    for (int i = 1; i < nblurs; i++) {
        // Swap input and output between blurs
        swap   = d_RIn;
        d_RIn  = d_ROut;
        d_ROut = swap;
        swap   = d_GIn;
        d_GIn  = d_GOut;
        d_GOut = swap;
        swap   = d_BIn;
        d_BIn  = d_BOut;
        d_BOut = swap;

        blurKernel<<<dimGrid, dimBlock>>>(*d_RIn, *d_GIn, *d_BIn, *d_ROut, *d_GOut, *d_BOut);
    }
    gettimeofday(&t5, NULL);
    copyMemoryFromDevice(*d_ROut, *d_GOut, *d_BOut, h_R, h_G, h_B);
    gettimeofday(&t6, NULL);
    freeDeviceMemory(*d_RIn, *d_GIn, *d_BIn, *d_ROut, *d_GOut, *d_BOut);
    gettimeofday(&t7, NULL);
    writeOutputFile(h_R, h_G, h_B, header, headerSize);
    gettimeofday(&t8, NULL);
    
    outputTimingResults(t1, t2, t3, t4, t5, t6, t7, t8);
}
